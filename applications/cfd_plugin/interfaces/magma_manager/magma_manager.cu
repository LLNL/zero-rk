#include "hip/hip_runtime.h"
#include "magma_manager.h"
#include "../../cuda_err_check.h"


template<typename T>
magma_manager<T>::magma_manager() :
  n_(-1),
  num_batches_(-1),
  factored_(false)
{
  magma_init();
  int device_id;
  magma_getdevice(&device_id);
  magma_queue_create(device_id, &magma_queue_);
}

template<typename T>
magma_manager<T>::~magma_manager()
{
  if(factored_) {
    FreeDeviceMemory();
  }
  magma_queue_destroy(magma_queue_);
  magma_finalize();
}

template<typename T>
void magma_manager<T>::setup_memory()
{
  if(factored_) {
    FreeDeviceMemory();
  }
  AllocateDeviceMemory();
}


template<typename T>
void magma_manager<T>::AllocateDeviceMemory()
{
  hipDeviceSynchronize();
  cuda_err_check(hipGetLastError());

  cuda_err_check(hipMalloc((void**)&matrix_inverse_dev_,sizeof(T)*(n_*n_*num_batches_)));
  cuda_err_check(hipMalloc((void**)&matrix_inverse_pointers_dev_,sizeof(T*)*num_batches_));
  cuda_err_check(hipMalloc((void**)&matrix_pointers_dev_,sizeof(T*)*num_batches_));
  cuda_err_check(hipMalloc((void**)&info_dev_,sizeof(int)*num_batches_));
  cuda_err_check(hipMalloc((void**)&ipiv_dev_,sizeof(int)*n_*num_batches_));
  cuda_err_check(hipMalloc((void**)&tmp_dev_,sizeof(T)*num_batches_*n_));
  cuda_err_check(hipMalloc((void**)&tmp_pointers_dev_,sizeof(T*)*num_batches_));
  cuda_err_check(hipMalloc((void**)&ipiv_pointers_dev_,sizeof(int*)*num_batches_));

  data_ptrs_.resize(num_batches_);

  std::vector<int*> tmpi_ptrs(num_batches_);
  std::vector<T*> tmp_ptrs(num_batches_);

  for(int j = 0; j < num_batches_; ++j) {
    data_ptrs_[j] = matrix_inverse_dev_ + j*n_*n_;
  }
  hipMemcpy(matrix_inverse_pointers_dev_, data_ptrs_.data(), sizeof(T*)*num_batches_, hipMemcpyHostToDevice);
  cuda_err_check(hipGetLastError());

  for(int j = 0; j < num_batches_; ++j) {
    tmp_ptrs[j] = tmp_dev_ + j*n_;
  }
  hipMemcpy(tmp_pointers_dev_, tmp_ptrs.data(), sizeof(T*)*num_batches_, hipMemcpyHostToDevice);
  cuda_err_check(hipGetLastError());

  for(int j = 0; j < num_batches_; ++j) {
    tmpi_ptrs[j] = ipiv_dev_ + j*n_;
  }
  hipMemcpy(ipiv_pointers_dev_, tmpi_ptrs.data(), sizeof(int*)*num_batches_, hipMemcpyHostToDevice);
  cuda_err_check(hipGetLastError());
}

template<typename T>
void magma_manager<T>::FreeDeviceMemory()
{
  hipFree(matrix_inverse_dev_);
  hipFree(matrix_inverse_pointers_dev_);
  hipFree(matrix_pointers_dev_);
  hipFree(info_dev_);
  hipFree(ipiv_dev_);
  hipFree(ipiv_pointers_dev_);
  hipFree(tmp_dev_);
  hipFree(tmp_pointers_dev_);
}

template<>
void magma_manager<double>::getrf_batched() {
  magma_dgetrf_batched(n_, /* number of rows per block */
                       n_, /* number of columns per block */
                       matrix_pointers_dev_,
                       n_, /* leading dimension of each block */
                       ipiv_pointers_dev_,
                       info_dev_,
                       num_batches_,
                       magma_queue_);
}

template<>
void magma_manager<hipDoubleComplex>::getrf_batched() {
  magma_zgetrf_batched(n_, /* number of rows per block */
                       n_, /* number of columns per block */
                       matrix_pointers_dev_,
                       n_, /* leading dimension of each block */
                       ipiv_pointers_dev_,
                       info_dev_,
                       num_batches_,
                       magma_queue_);
}

template<>
void magma_manager<double>::getri_batched() {
  magma_dgetri_outofplace_batched(n_, /* order of block */
                                  matrix_pointers_dev_,
                                  n_, /* leading dimension of each block */
                                  ipiv_pointers_dev_,
                                  matrix_inverse_pointers_dev_,
                                  n_, /* leading dimension of each block of inverse */
                                  info_dev_,
                                  num_batches_,
                                  magma_queue_);
}

template<>
void magma_manager<hipDoubleComplex>::getri_batched() {
  magma_zgetri_outofplace_batched(n_, /* order of block */
                                  matrix_pointers_dev_,
                                  n_, /* leading dimension of each block */
                                  ipiv_pointers_dev_,
                                  matrix_inverse_pointers_dev_,
                                  n_, /* leading dimension of each block of inverse */
                                  info_dev_,
                                  num_batches_,
                                  magma_queue_);
}


template<typename T>
int magma_manager<T>::factor_invert(int num_batches, int n, T* values) {
  if(n != n_ || num_batches != num_batches_) {
    n_ = n;
    num_batches_ = num_batches;
    setup_memory();
  }
  if(values == NULL) {
    return 1;
  }

  bool need_tx = false;
  for(int j = 0; j < num_batches_; ++j) {
    if(data_ptrs_[j] != values + j*n_*n_) {
      data_ptrs_[j] = values + j*n_*n_;
      need_tx = true;
    }
  }
  if(need_tx) {
    hipMemcpy(matrix_pointers_dev_, data_ptrs_.data(), sizeof(T*)*num_batches_, hipMemcpyHostToDevice);
  }

  getrf_batched();
  getri_batched();


  int ierr = 0;
#ifdef ZERORK_FULL_DEBUG
  info_.resize(num_batches_);
  cuda_err_check(hipMemcpy(info_.data(), info_dev_, num_batches_*sizeof(int), hipMemcpyDeviceToHost));
  //Check for errors
  // factor_error > 0, singular matrix, zero diagonal at row,col = factor_error
  // factor_error = 0, success
  // factor_error < 0, illegal input
  for(int i=0; i < num_batches_; ++i) {
    if(info_[i]!=0) {
      ierr = info_[i];
      break;
    }
  }
#endif

  factored_ = true;
  return ierr;
}

template<typename T>
int magma_manager<T>::factor_lu(int num_batches, int n, T* values) {
  if(n != n_ || num_batches != num_batches_) {
    n_ = n;
    num_batches_ = num_batches;
    setup_memory();
  }
  if(values == NULL) {
    return 1;
  }

  bool need_tx = false;
  for(int j = 0; j < num_batches_; ++j) {
    if(data_ptrs_[j] != values + j*n_*n_) {
      data_ptrs_[j] = values + j*n_*n_;
      need_tx = true;
    }
  }
  if(need_tx) {
    hipMemcpy(matrix_pointers_dev_, data_ptrs_.data(), sizeof(T*)*num_batches_, hipMemcpyHostToDevice);
  }

  getrf_batched();

  int ierr = 0;
#ifdef ZERORK_FULL_DEBUG
  info_.resize(num_batches_);
  cuda_err_check(hipMemcpy(info_.data(), info_dev_, num_batches_*sizeof(int), hipMemcpyDeviceToHost));
  //Check for errors
  // factor_error > 0, singular matrix, zero diagonal at row,col = factor_error
  // factor_error = 0, success
  // factor_error < 0, illegal input
  for(int i=0; i < num_batches_; ++i) {
    if(info_[i]!=0) {
      ierr = info_[i];
      break;
    }
  }
#endif

  factored_ = true;
  return ierr;
}


//The following modified from cuda sdk-5.0
#define TRANSPOSE_TILE_DIM    32
#define TRANSPOSE_BLOCK_ROWS  8

template<typename T>
static __global__ void MAGMA_MANAGER_TransposeNoBankConflicts(T *odata, const T *idata, const int width, const int height)
{
    __shared__ T tile[TRANSPOSE_TILE_DIM][TRANSPOSE_TILE_DIM+1];
    int xIndex,yIndex,index_in,index_out;

    xIndex = blockIdx.x * TRANSPOSE_TILE_DIM + threadIdx.x;
    yIndex = blockIdx.y * TRANSPOSE_TILE_DIM + threadIdx.y;
    index_in = xIndex + (yIndex)*width;

    for (int i=0; i<TRANSPOSE_TILE_DIM; i+=TRANSPOSE_BLOCK_ROWS)
    {
        if(xIndex < width && yIndex+i < height){
        tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];}
    }

    __syncthreads();

    xIndex = blockIdx.y * TRANSPOSE_TILE_DIM + threadIdx.x;
    yIndex = blockIdx.x * TRANSPOSE_TILE_DIM + threadIdx.y;
    index_out = xIndex + (yIndex)*height;

    for (int i=0; i<TRANSPOSE_TILE_DIM; i+=TRANSPOSE_BLOCK_ROWS)
    {
        if(yIndex+i < width && xIndex < height){
        odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];}
    }
}

template<typename T>
void magma_manager<T>::cuda_transpose(T* odata, const T* idata, const int width, const int height)
{
    // Put df/dy in "normal" order
    dim3 nBlocks2D,nThreads2D;
    nThreads2D.x = TRANSPOSE_TILE_DIM;
    nThreads2D.y = TRANSPOSE_BLOCK_ROWS;
    nBlocks2D.x = (width+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM;
    nBlocks2D.y = (height+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM;
    MAGMA_MANAGER_TransposeNoBankConflicts<T><<<nBlocks2D,nThreads2D>>>(odata,idata,width,height);
#ifdef ZERORK_FULL_DEBUG
    cuda_err_check( hipPeekAtLastError() );
    cuda_err_check( hipDeviceSynchronize() );
#endif
}


namespace {
template<typename T>
void __global__ MAGMA_MANAGER_cuda_bdmv_kernel
(
    const int mtx_block_size,
    const int num_mtx_blocks,
    const T* A_dev,
    const T* X_dev ,
    T * Y_dev
)
{
  int tidx = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  for( ; tidx < num_mtx_blocks*mtx_block_size; tidx += stride)
  {
    int local_row   = tidx % mtx_block_size;
    int local_block = tidx / mtx_block_size;
    T Y_dev_accum = 0.0;
    for(int i = 0; i < mtx_block_size; ++i) //columns
    {
      int data_idx = mtx_block_size*mtx_block_size*local_block + mtx_block_size*i + local_row;
      Y_dev_accum += A_dev[data_idx]*X_dev[i+local_block*mtx_block_size];
    }
    Y_dev[local_row+local_block*mtx_block_size] = Y_dev_accum;
  }
}

template<>
void __global__ MAGMA_MANAGER_cuda_bdmv_kernel
(
    const int mtx_block_size,
    const int num_mtx_blocks,
    const hipDoubleComplex* A_dev,
    const hipDoubleComplex* X_dev ,
    hipDoubleComplex * Y_dev
)
{
  int tidx = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  for( ; tidx < num_mtx_blocks*mtx_block_size; tidx += stride)
  {
    int local_row   = tidx % mtx_block_size;
    int local_block = tidx / mtx_block_size;
    hipDoubleComplex Y_dev_accum = make_hipDoubleComplex(0.0,0.0);
    for(int i = 0; i < mtx_block_size; ++i) //columns
    {
      int data_idx = mtx_block_size*mtx_block_size*local_block + mtx_block_size*i + local_row;
      //Y_dev_accum += A_dev[data_idx]*X_dev[i+local_block*mtx_block_size];
      Y_dev_accum = hipCadd(Y_dev_accum, hipCmul(A_dev[data_idx],X_dev[i+local_block*mtx_block_size]));
    }
    Y_dev[local_row+local_block*mtx_block_size] = Y_dev_accum;
  }
}
} //anonymous namespace

template<typename T>
int magma_manager<T>::cuda_bdmv(int n, int nbatch, T* A_dev, T* B_dev, T* Y_dev)
{
  int threads = std::min(n*nbatch,1024);
  int blocks=(nbatch*n+threads-1)/threads;
  MAGMA_MANAGER_cuda_bdmv_kernel<T><<<blocks,threads>>>(n, nbatch, A_dev, B_dev, Y_dev);
#ifdef ZERORK_FULL_DEBUG
  cuda_err_check(hipPeekAtLastError());
  cuda_err_check(hipDeviceSynchronize());
#endif
  return 0;  
}

template<typename T>
int magma_manager<T>::solve_invert(int num_batches, int n, const T* rhs, T* soln) {
  if(n != n_ || num_batches != num_batches_) {
    return 1;
  }

  // Transpose rhs into soln
  cuda_transpose(soln,rhs,num_batches_,n_);

  // Block-diagonal matrix vector multiplication
  cuda_bdmv(n_, num_batches_, matrix_inverse_dev_, soln, tmp_dev_);

  // Put tmp back into block order
  cuda_transpose(soln,tmp_dev_,n_,num_batches_);

  return(0);
}

template<>
void magma_manager<double>::getrs_batched() {
  // Magma forward and back substitution
  magma_dgetrs_batched(MagmaNoTrans,
                       n_, /* order of the matrix */
                       1, /* number of right hand sides */
                       matrix_pointers_dev_,
                       n_, /* leading dimension of A */
                       ipiv_pointers_dev_,
                       tmp_pointers_dev_, /* right hand side (input), solution (output) */
                       n_, /* leading dimension of b */
                       num_batches_,
                       magma_queue_);
}

template<>
void magma_manager<hipDoubleComplex>::getrs_batched() {
  // Magma forward and back substitution
  magma_zgetrs_batched(MagmaNoTrans,
                       n_, /* order of the matrix */
                       1, /* number of right hand sides */
                       matrix_pointers_dev_,
                       n_, /* leading dimension of A */
                       ipiv_pointers_dev_,
                       tmp_pointers_dev_, /* right hand side (input), solution (output) */
                       n_, /* leading dimension of b */
                       num_batches_,
                       magma_queue_);
}

template<typename T>
int magma_manager<T>::solve_lu(int num_batches, int n, const T* rhs, T* soln) {
  if(n != n_ || num_batches != num_batches_) {
    return 1;
  }

  // Transpose rhs into tmp_dev_
  cuda_transpose(tmp_dev_,rhs,num_batches_,n_);

  getrs_batched();

  // Put tmp back into block order
  cuda_transpose(soln,tmp_dev_,n_,num_batches_);

  return(0);
}

template class magma_manager<double>;
template class magma_manager<hipDoubleComplex>;

