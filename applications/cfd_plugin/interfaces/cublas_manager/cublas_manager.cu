#include "hip/hip_runtime.h"
#include "cublas_manager.h"
#include "../../cuda_err_check.h"


cublas_manager::cublas_manager() :
  n_(-1),
  num_batches_(-1),
  factored_(false)
{
  hipblasCreate(&cublas_handle_);
}

cublas_manager::~cublas_manager()
{
  if(factored_) {
    FreeDeviceMemory();
    hipblasDestroy(cublas_handle_);
  }
}

void cublas_manager::setup_memory()
{
  if(factored_) {
    FreeDeviceMemory();
  }
  AllocateDeviceMemory();
}


void cublas_manager::AllocateDeviceMemory()
{
  hipDeviceSynchronize();
  cuda_err_check(hipGetLastError());

  cuda_err_check(hipMalloc((void**)&matrix_inverse_dev_,sizeof(double)*(n_*n_*num_batches_)));
  cuda_err_check(hipMalloc((void**)&matrix_inverse_pointers_dev_,sizeof(double*)*num_batches_));
  cuda_err_check(hipMalloc((void**)&matrix_pointers_dev_,sizeof(double*)*num_batches_));
  cuda_err_check(hipMalloc((void**)&info_dev_,sizeof(int)*num_batches_));
  cuda_err_check(hipMalloc((void**)&tmp_dev_,sizeof(double)*num_batches_*n_));
  cuda_err_check(hipMalloc((void**)&tmp_pointers_dev_,sizeof(double*)*num_batches_));

  data_ptrs_.resize(num_batches_);
  tmp_ptrs_.resize(num_batches_);

  for(int j = 0; j < num_batches_; ++j) {
    data_ptrs_[j] = matrix_inverse_dev_ + j*n_*n_;
  }
  hipMemcpy(matrix_inverse_pointers_dev_, data_ptrs_.data(), sizeof(double*)*num_batches_, hipMemcpyHostToDevice);
  cuda_err_check(hipGetLastError());

  for(int j = 0; j < num_batches_; ++j) {
    tmp_ptrs_[j] = tmp_dev_ + j*n_;
  }
  hipMemcpy(tmp_pointers_dev_, tmp_ptrs_.data(), sizeof(double*)*num_batches_, hipMemcpyHostToDevice);
  cuda_err_check(hipGetLastError());
}

void cublas_manager::FreeDeviceMemory()
{
  hipFree(matrix_inverse_dev_);
  hipFree(matrix_inverse_pointers_dev_);
  hipFree(matrix_pointers_dev_);
  hipFree(info_dev_);
  hipFree(tmp_dev_);
  hipFree(tmp_pointers_dev_);
}

int cublas_manager::factor_invert(int num_batches, int n, double* values) {
  if(n != n_ || num_batches != num_batches_) {
    n_ = n;
    num_batches_ = num_batches;
    setup_memory();
  }
  if(values == NULL) {
    return 1;
  }

  bool need_tx = false;
  for(int j = 0; j < num_batches_; ++j) {
    if(data_ptrs_[j] != values + j*n_*n_) {
      data_ptrs_[j] = values + j*n_*n_;
      need_tx = true;
    }
  }
  if(need_tx) {
    hipMemcpy(matrix_pointers_dev_, data_ptrs_.data(), sizeof(double*)*num_batches_, hipMemcpyHostToDevice);
  }

  int lda = n_;
  int* ipiv = NULL; //Turns off pivoting
  hipblasDgetrfBatched(cublas_handle_, n_,
                      matrix_pointers_dev_, lda,
                      ipiv, info_dev_, num_batches_);

  int ldc = n_;
  const double** const_matrix_pointers_dev = (const double**) matrix_pointers_dev_;
  hipblasDgetriBatched(cublas_handle_, n_, const_matrix_pointers_dev,
                      lda, ipiv, matrix_inverse_pointers_dev_,
                      ldc, info_dev_, num_batches_);

  int ierr = 0;
#ifdef ZERORK_FULL_DEBUG
  info_.resize(num_batches_);
  cuda_err_check(hipMemcpy(info_.data(), info_dev_, num_batches_*sizeof(int), hipMemcpyDeviceToHost));
  //Check for errors
  // factor_error > 0, singular matrix, zero diagonal at row,col = factor_error
  // factor_error = 0, success
  // factor_error < 0, illegal input
  for(int i=0; i < num_batches_; ++i) {
    if(info_[i]!=0) {
      ierr = info_[i];
      break;
    }
  }
#endif

  factored_ = true;
  return ierr;
}

int cublas_manager::factor_lu(int num_batches, int n, double* values) {
  if(n != n_ || num_batches != num_batches_) {
    n_ = n;
    num_batches_ = num_batches;
    setup_memory();
  }
  if(values == NULL) {
    return 1;
  }

  bool need_tx = false;
  for(int j = 0; j < num_batches_; ++j) {
    if(data_ptrs_[j] != values + j*n_*n_) {
      data_ptrs_[j] = values + j*n_*n_;
      need_tx = true;
    }
  }
  if(need_tx) {
    hipMemcpy(matrix_pointers_dev_, data_ptrs_.data(), sizeof(double*)*num_batches_, hipMemcpyHostToDevice);
  }

  int lda = n_;
  int* ipiv = NULL; //Turns off pivoting
  hipblasDgetrfBatched(cublas_handle_, n_,
                      matrix_pointers_dev_, lda,
                      ipiv, info_dev_, num_batches_);

  int ierr = 0;
#ifdef ZERORK_FULL_DEBUG
  info_.resize(num_batches_);
  cuda_err_check(hipMemcpy(info_.data(), info_dev_, num_batches_*sizeof(int), hipMemcpyDeviceToHost));
  //Check for errors
  // factor_error > 0, singular matrix, zero diagonal at row,col = factor_error
  // factor_error = 0, success
  // factor_error < 0, illegal input
  for(int i=0; i < num_batches_; ++i) {
    if(info_[i]!=0) {
      ierr = info_[i];
      break;
    }
  }
#endif

  factored_ = true;
  return ierr;
}


//The following modified from cuda sdk-5.0
#define TRANSPOSE_TILE_DIM    32
#define TRANSPOSE_BLOCK_ROWS  8

static __global__ void CUBLAS_MANAGER_TransposeNoBankConflicts(double *odata, const double *idata, const int width, const int height)
{
    __shared__ double tile[TRANSPOSE_TILE_DIM][TRANSPOSE_TILE_DIM+1];
    int xIndex,yIndex,index_in,index_out;

    xIndex = blockIdx.x * TRANSPOSE_TILE_DIM + threadIdx.x;
    yIndex = blockIdx.y * TRANSPOSE_TILE_DIM + threadIdx.y;
    index_in = xIndex + (yIndex)*width;

    for (int i=0; i<TRANSPOSE_TILE_DIM; i+=TRANSPOSE_BLOCK_ROWS)
    {
        if(xIndex < width && yIndex+i < height){
        tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];}
    }

    __syncthreads();

    xIndex = blockIdx.y * TRANSPOSE_TILE_DIM + threadIdx.x;
    yIndex = blockIdx.x * TRANSPOSE_TILE_DIM + threadIdx.y;
    index_out = xIndex + (yIndex)*height;

    for (int i=0; i<TRANSPOSE_TILE_DIM; i+=TRANSPOSE_BLOCK_ROWS)
    {
        if(yIndex+i < width && xIndex < height){
        odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];}
    }
}

void cublas_manager::cuda_transpose(double* odata, const double* idata, const int width, const int height)
{
    // Put df/dy in "normal" order
    dim3 nBlocks2D,nThreads2D;
    nThreads2D.x = TRANSPOSE_TILE_DIM;
    nThreads2D.y = TRANSPOSE_BLOCK_ROWS;
    nBlocks2D.x = (width+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM;
    nBlocks2D.y = (height+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM;
    CUBLAS_MANAGER_TransposeNoBankConflicts<<<nBlocks2D,nThreads2D>>>(odata,idata,width,height);
#ifdef ZERORK_FULL_DEBUG
    cuda_err_check( hipPeekAtLastError() );
    cuda_err_check( hipDeviceSynchronize() );
#endif
}


static void __global__ CUBLAS_MANAGER_cuda_bdmv_kernel
(
    const int mtx_block_size,
    const int num_mtx_blocks,
    const double* A_dev,
    const double* X_dev ,
    double * Y_dev
)
{
  int tidx = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  for( ; tidx < num_mtx_blocks*mtx_block_size; tidx += stride)
  {
    int local_row   = tidx % mtx_block_size;
    int local_block = tidx / mtx_block_size;
    double Y_dev_accum = 0.0;
    for(int i = 0; i < mtx_block_size; ++i) //columns
    {
      int data_idx = mtx_block_size*mtx_block_size*local_block + mtx_block_size*i + local_row;
      Y_dev_accum += A_dev[data_idx]*X_dev[i+local_block*mtx_block_size];
    }
    Y_dev[local_row+local_block*mtx_block_size] = Y_dev_accum;
  }
}

int cublas_manager::cuda_bdmv(int n, int nbatch, double* A_dev, double* B_dev, double* Y_dev)
{
  int threads = std::min(n*nbatch,1024);
  int blocks=(nbatch*n+threads-1)/threads;
  CUBLAS_MANAGER_cuda_bdmv_kernel<<<blocks,threads>>>(n, nbatch, A_dev, B_dev, Y_dev);
#ifdef ZERORK_FULL_DEBUG
  cuda_err_check(hipPeekAtLastError());
  cuda_err_check(hipDeviceSynchronize());
#endif
  return 0;  
}

int cublas_manager::solve_invert(int num_batches, int n, const double* rhs, double* soln) {
  if(n != n_ || num_batches != num_batches_) {
    return 1;
  }

  // Transpose rhs into soln
  cuda_transpose(soln,rhs,num_batches_,n_);

  // Block-diagonal matrix vector multiplication
  cuda_bdmv(n_, num_batches_, matrix_inverse_dev_, soln, tmp_dev_);

  // Put tmp back into block order
  cuda_transpose(soln,tmp_dev_,n_,num_batches_);

  return(0);
}

int cublas_manager::solve_lu(int num_batches, int n, const double* rhs, double* soln) {
  if(n != n_ || num_batches != num_batches_) {
    return 1;
  }

  // Transpose rhs into tmp_dev_
  cuda_transpose(tmp_dev_,rhs,num_batches_,n_);

  // CUBLAS forward and back substitution
  int* ipiv = NULL; //Turns off pivoting
  int lda = n_;
  int ldb = n_;
  int info = 0;
  hipblasDgetrsBatched(cublas_handle_, HIPBLAS_OP_N, n_, 1,
                      matrix_pointers_dev_, lda,
                      ipiv, tmp_pointers_dev_, ldb, &info, num_batches_);

  // Put tmp back into block order
  cuda_transpose(soln,tmp_dev_,n_,num_batches_);

  return(0);
}

